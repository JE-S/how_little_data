#include "hip/hip_runtime.h"
// repelacing sys. mat. storage
// loads in whole sino for POCS and bp
#ifndef max
	#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
//const unsigned int blocksize = 256;
//const unsigned int nblocks = 4;
//unsigned int nsimulrays = blocksize*nblocks;
const int nframecomp=8;

__global__ void multiraybp(int* intparms, float* floatparms,
		   float* sinovals, char* sinoinds, float* smat);


__global__ void multirayproj(int* intparms, float* floatparms,
                   float* sinovals, char* sinoinds, float* smat);



extern "C" void backproject(
   float* sinomat, char* indsino,
   float* frame_vectors,
   int ns, int nu,
   float du,float u0,
   float* smat,
   float dx, float dy,
   float x0,float y0,
   int nx,int ny,
   unsigned int nblocks, unsigned int blocksize);


extern "C" void  rayproj(
   float* sinomat, char* indsino,
   float* frame_vectors,
   int ns, int nu,
   float du,float u0,
   float* smat,
   float dx, float dy,
   float x0,float y0,
   int nx,int ny,
   unsigned int nblocks, unsigned int blocksize);





extern "C" void setGPUdevice(int devnum){
  hipSetDevice(devnum);
}
extern "C" int getGPUdevice(){
  int devnum;
  hipGetDevice(&devnum);
  return devnum;
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





void backproject(
   float* sinomat, char* indsino,
   float* frame_vectors,
   int ns, int nu,
   float du,float u0,
   float* smat,
   float dx, float dy,
   float x0,float y0,
   int nx,int ny,
   unsigned int nblocks, unsigned int blocksize) {


  unsigned int nsimulrays = nblocks*blocksize;

  float xSourceCenter,ySourceCenter,xDetCenter, yDetCenter, eux, euy;

  //  float val;

  int ip;

  int ngroups = (nu+nsimulrays-1)/nsimulrays;



  float* floatparms;
  int* intparms;

  floatparms = (float *)malloc(13*sizeof(float));
  intparms = (int *)malloc(5*sizeof(int));


  size_t imsize = nx*ny*sizeof(float);
  float* d_smat;
  hipMalloc((void**)&d_smat,imsize);

  size_t sinosize = nu*sizeof(float);
  float* d_sinovals;
  hipMalloc((void**)&d_sinovals,sinosize);

  size_t indsize = nu*sizeof(char);
  char* d_sinoinds;
  hipMalloc((void**)&d_sinoinds,indsize);

  size_t ipsize = 5*sizeof(int);
  int* d_intparms;
  hipMalloc((void**)&d_intparms,ipsize);

  size_t dpsize = 13*sizeof(float);
  float* d_floatparms;
  hipMalloc((void**)&d_floatparms,dpsize);


  checkErrors("memory allocation");

  intparms[0] = 0;
  intparms[1] = ngroups;
  intparms[2] = nx; intparms[3] = ny; intparms[4] = nu;

  floatparms[0] = 0.;
  floatparms[1] = x0; floatparms[2] = y0; floatparms[3] = dx; floatparms[4] = dy;
  floatparms[5] = du; floatparms[6] = u0;

  hipMemcpy(d_smat,smat,imsize,hipMemcpyHostToDevice);
  for (ip=0; ip<ns; ip++){

     xSourceCenter=frame_vectors[ip*nframecomp+ 0];
     ySourceCenter=frame_vectors[ip*nframecomp+ 1];

     xDetCenter=frame_vectors[ip*nframecomp+ 2];
     yDetCenter=frame_vectors[ip*nframecomp+ 3];

     eux=frame_vectors[ip*nframecomp+ 4];
     euy=frame_vectors[ip*nframecomp+ 5];

     floatparms[7] = xSourceCenter; floatparms[8] = ySourceCenter;
     floatparms[9] = xDetCenter; floatparms[10] = yDetCenter;
     floatparms[11] = eux; floatparms[12] = euy;



     hipMemcpy(d_floatparms,floatparms,dpsize,hipMemcpyHostToDevice);

       hipMemcpy(d_intparms,intparms,ipsize,hipMemcpyHostToDevice);
       hipMemcpy(d_sinovals,sinomat+ip*nu,sinosize,hipMemcpyHostToDevice);
       hipMemcpy(d_sinoinds,indsino+ip*nu,indsize,hipMemcpyHostToDevice);
       checkErrors("copy data to device");

       multiraybp<<<nblocks,blocksize>>>(d_intparms,d_floatparms,
	            d_sinovals,d_sinoinds,d_smat);
       checkErrors("compute on device");
  }
  hipMemcpy(smat,d_smat,imsize,hipMemcpyDeviceToHost);
  checkErrors("copy data from device");
  free(intparms);
  free(floatparms);
  hipFree(d_smat);
  hipFree(d_sinovals);
  hipFree(d_sinoinds);
  hipFree(d_intparms);
  hipFree(d_floatparms);
}





void  rayproj(
   float* sinomat, char* indsino,
   float* frame_vectors,
   int ns, int nu,
   float du,float u0,
   float* smat,
   float dx, float dy,
   float x0,float y0,
   int nx,int ny,
   unsigned int nblocks, unsigned int blocksize) {


  float xSourceCenter,ySourceCenter,xDetCenter, yDetCenter, eux, euy;
  unsigned int nsimulrays = nblocks*blocksize;

  //  float val;

  int ip;

  int ngroups = (nu+nsimulrays-1)/nsimulrays;



  float* floatparms;
  int* intparms;

  floatparms = (float *)malloc(13*sizeof(float));
  intparms = (int *)malloc(5*sizeof(int));


  size_t imsize = nx*ny*sizeof(float);
  float* d_smat;
  hipMalloc((void**)&d_smat,imsize);

  size_t sinosize = nu*sizeof(float);
  float* d_sinovals;
  hipMalloc((void**)&d_sinovals,sinosize);

  size_t indsize = nu*sizeof(char);
  char* d_sinoinds;
  hipMalloc((void**)&d_sinoinds,indsize);

  size_t ipsize = 5*sizeof(int);
  int* d_intparms;
  hipMalloc((void**)&d_intparms,ipsize);

  size_t dpsize = 13*sizeof(float);
  float* d_floatparms;
  hipMalloc((void**)&d_floatparms,dpsize);



  checkErrors("memory allocation");

  intparms[0] = 0;
  intparms[1] = ngroups;
  intparms[2] = nx; intparms[3] = ny; intparms[4] = nu;

  floatparms[0] = 0.;
  floatparms[1] = x0; floatparms[2] = y0; floatparms[3] = dx; floatparms[4] = dy;
  floatparms[5] = du; floatparms[6] = u0;

  hipMemcpy(d_smat,smat,imsize,hipMemcpyHostToDevice);
  for (ip=0; ip<ns; ip++){

     xSourceCenter=frame_vectors[ip*nframecomp+ 0];
     ySourceCenter=frame_vectors[ip*nframecomp+ 1];

     xDetCenter=frame_vectors[ip*nframecomp+ 2];
     yDetCenter=frame_vectors[ip*nframecomp+ 3];

     eux=frame_vectors[ip*nframecomp+ 4];
     euy=frame_vectors[ip*nframecomp+ 5];

     floatparms[7] = xSourceCenter; floatparms[8] = ySourceCenter;
     floatparms[9] = xDetCenter; floatparms[10] = yDetCenter;
     floatparms[11] = eux; floatparms[12] = euy;


  

     hipMemcpy(d_floatparms,floatparms,dpsize,hipMemcpyHostToDevice);

       hipMemcpy(d_intparms,intparms,ipsize,hipMemcpyHostToDevice);
       hipMemcpy(d_sinovals,sinomat+ip*nu,sinosize,hipMemcpyHostToDevice);
       hipMemcpy(d_sinoinds,indsino+ip*nu,indsize,hipMemcpyHostToDevice);
       checkErrors("copy data to device");

       multirayproj<<<nblocks,blocksize>>>(d_intparms,d_floatparms,
	            d_sinovals,d_sinoinds,d_smat);
       checkErrors("compute on device");

       hipMemcpy(sinomat+ip*nu,d_sinovals,sinosize,hipMemcpyDeviceToHost);
       checkErrors("copy data from device");
      


  }

  free(intparms);
  free(floatparms);
  hipFree(d_smat);
  hipFree(d_sinovals);
  hipFree(d_sinoinds);
  hipFree(d_intparms);
  hipFree(d_floatparms);
}





__global__ void multiraybp(int* intparms, float* floatparms,
                   float* sinovals, char* sinoinds, float* smat){




__shared__ int ngroups, nx,ny,nu;
__shared__ float x0,y0,dx,dy,du,u0;
__shared__ float xSourceCenter,ySourceCenter,xDetCenter,yDetCenter,eux,euy;




ngroups=intparms[1];
nx=intparms[2]; ny=intparms[3]; nu=intparms[4];

x0=floatparms[1]; y0=floatparms[2]; dx=floatparms[3]; dy=floatparms[4];
du=floatparms[5]; u0=floatparms[6];

xSourceCenter=floatparms[7]; ySourceCenter=floatparms[8];
xDetCenter=floatparms[9]; yDetCenter=floatparms[10];
eux=floatparms[11]; euy=floatparms[12];


int ng, iyOld, ixOld, ix, iy, jp, jpp;
float u, xbin, ybin, xsource, ysource, xl, yl, xdiff, ydiff, xad, yad, slope, slopeinv, x, y;
float travPixlen, yMid, xMid;
float yIntercept, yIntOld, ydist1, ydist2, frac1, frac2;
float xIntercept, xIntOld, xdist1, xdist2;
float val0;

jpp =  blockDim.x*blockIdx.x + threadIdx.x;
for(ng=0;ng<ngroups;ng++){
jp = ng + ngroups*jpp;
if ((jp<nu) && (sinoinds[jp] == 1)){
   val0 = sinovals[jp];

   u = u0+(jp+0.5f)*du;
   xbin = xDetCenter + eux*u;
   ybin = yDetCenter + euy*u;
   xsource = xSourceCenter;
   ysource = ySourceCenter;


   xl=x0;
   yl=y0;

   xdiff=xbin-xsource;
   ydiff=ybin-ysource;
   xad=fabs(xdiff)*dy;
   yad=fabs(ydiff)*dx;

   if (xad>yad) {
      slope=ydiff/xdiff;
      travPixlen=dx*sqrt(1.0f+slope*slope);
      yIntOld=ysource+slope*(xl-xsource);
      iyOld=floor((yIntOld-y0)/dy);
           for (ix=0; ix<nx; ix++) {
	     x=xl+dx*(ix + 1.0f);
	     yIntercept=ysource+slope*(x-xsource);
	     iy=floor((yIntercept-y0)/dy);
	     if (iy == iyOld) {
	       if ((iy >= 0) && (iy < ny)) {
                 smat[ix*ny + iy] = smat[ix*ny + iy]+
	            val0*travPixlen;
		       }
		    }else{
	       yMid=dy*max(iy,iyOld)+yl;
	       ydist1=fabs(yMid-yIntOld);
	       ydist2=fabs(yIntercept-yMid);
	       frac1=ydist1/(ydist1+ydist2);
	       frac2=1.0f-frac1;
	       if ((iyOld >= 0) && (iyOld < ny)) {
                 smat[ix*ny + iyOld] = smat[ix*ny + iyOld]+
	            val0*frac1*travPixlen;
		       }
	       if ((iy>=0) && (iy<ny)) {
                 smat[ix*ny + iy] = smat[ix*ny + iy]+
	            val0*frac2*travPixlen;
		       }
		    }
	     iyOld=iy;
	     yIntOld=yIntercept;
	    }
		    }else{


	   slopeinv=xdiff/ydiff;
	   travPixlen=dy*sqrt(1.0f+slopeinv*slopeinv);
	   xIntOld=xsource+slopeinv*(yl-ysource);
	   ixOld=floor((xIntOld-x0)/dx);
           for (iy=0; iy<ny; iy++) {
	     y=yl+dy*(iy + 1.0f);
	     xIntercept=xsource+slopeinv*(y-ysource);
	     ix=floor((xIntercept-x0)/dx);
	     if (ix == ixOld) {
	       if ((ix >= 0) && (ix < nx)) {
                 smat[ix*ny + iy] = smat[ix*ny + iy]+
	            val0*travPixlen;
		       }
		    }else{
	       xMid=dx*max(ix,ixOld)+xl;
	       xdist1=fabs(xMid-xIntOld);
	       xdist2=fabs(xIntercept-xMid);
	       frac1=xdist1/(xdist1+xdist2);
	       frac2=1.0f-frac1;
	       if ((ixOld >= 0) && (ixOld < nx)) {
                 smat[ixOld*ny + iy] = smat[ixOld*ny + iy]+
	            val0*frac1*travPixlen;
		       }
	       if ((ix>=0) && (ix<nx)) {
                 smat[ix*ny + iy] = smat[ix*ny + iy]+
	            val0*frac2*travPixlen;
		       }
		    }
	     ixOld=ix;
	     xIntOld=xIntercept;
	    }


		    }




	}
__syncthreads();
}

}




__global__ void multirayproj(int* intparms, float* floatparms,
                   float* sinovals, char* sinoinds, float* smat){




__shared__ int ngroups, nx,ny,nu;
__shared__ float x0,y0,dx,dy,du,u0;
__shared__ float xSourceCenter,ySourceCenter,xDetCenter,yDetCenter,eux,euy;




ngroups=intparms[1];
nx=intparms[2]; ny=intparms[3]; nu=intparms[4];

x0=floatparms[1]; y0=floatparms[2]; dx=floatparms[3]; dy=floatparms[4];
du=floatparms[5]; u0=floatparms[6];

xSourceCenter=floatparms[7]; ySourceCenter=floatparms[8];
xDetCenter=floatparms[9]; yDetCenter=floatparms[10];
eux=floatparms[11]; euy=floatparms[12];


int ng,iyOld, ixOld, ix, iy, jp, jpp;
float u, xbin, ybin, xsource, ysource, xl, yl, xdiff, ydiff, xad, yad, slope, slopeinv, x, y;
float travPixlen, raysum, yMid, xMid;
float yIntercept, yIntOld, ydist1, ydist2, frac1, frac2;
float xIntercept, xIntOld, xdist1, xdist2;

jpp =  blockDim.x*blockIdx.x + threadIdx.x;

for(ng=0; ng<ngroups; ng++){
jp = ng + ngroups*jpp;
if ((jp<nu) && (sinoinds[jp] == 1)){

   u = u0+(jp+0.5f)*du;
   xbin = xDetCenter + eux*u;
   ybin = yDetCenter + euy*u;
   xsource = xSourceCenter;
   ysource = ySourceCenter;


   xl=x0;
   yl=y0;

   xdiff=xbin-xsource;
   ydiff=ybin-ysource;
   xad=fabs(xdiff)*dy;
   yad=fabs(ydiff)*dx;

   if (xad>yad) {
      slope=ydiff/xdiff;
      travPixlen=dx*sqrt(1.0f+slope*slope);
      yIntOld=ysource+slope*(xl-xsource);
      iyOld=floor((yIntOld-y0)/dy);
      raysum=0.;
           for (ix=0; ix<nx; ix++) {
	     x=xl+dx*(ix + 1.0f);
	     yIntercept=ysource+slope*(x-xsource);
	     iy=floor((yIntercept-y0)/dy);
	     if (iy == iyOld) {
	       if ((iy >= 0) && (iy < ny)) {
		 raysum=raysum+travPixlen*smat[ix*ny + iy];
		       }
		    }else{
	       yMid=dy*max(iy,iyOld)+yl;
	       ydist1=fabs(yMid-yIntOld);
	       ydist2=fabs(yIntercept-yMid);
	       frac1=ydist1/(ydist1+ydist2);
	       frac2=1.0f-frac1;
	       if ((iyOld >= 0) && (iyOld < ny)) {
		 raysum=raysum+frac1*travPixlen*smat[ix*ny + iyOld];
		       }
	       if ((iy>=0) && (iy<ny)) {
		 raysum=raysum+frac2*travPixlen*smat[ix*ny + iy];
		       }
		    }
	     iyOld=iy;
	     yIntOld=yIntercept;
	    }
		    }else{


	   slopeinv=xdiff/ydiff;
	   travPixlen=dy*sqrt(1.0f+slopeinv*slopeinv);
	   xIntOld=xsource+slopeinv*(yl-ysource);
	   ixOld=floor((xIntOld-x0)/dx);
	   raysum=0.f;
           for (iy=0; iy<ny; iy++) {
	     y=yl+dy*(iy + 1.0f);
	     xIntercept=xsource+slopeinv*(y-ysource);
	     ix=floor((xIntercept-x0)/dx);
	     if (ix == ixOld) {
	       if ((ix >= 0) && (ix < nx)) {
		 raysum=raysum+travPixlen*smat[ix*ny + iy];
		       }
		    }else{
	       xMid=dx*max(ix,ixOld)+xl;
	       xdist1=fabs(xMid-xIntOld);
	       xdist2=fabs(xIntercept-xMid);
	       frac1=xdist1/(xdist1+xdist2);
	       frac2=1.0f-frac1;
	       if ((ixOld >= 0) && (ixOld < nx)) {
		 raysum=raysum+frac1*travPixlen*smat[ixOld*ny + iy];
		       }
	       if ((ix>=0) && (ix<nx)) {
                 raysum=raysum+frac2*travPixlen*smat[ix*ny + iy];
		       }
		    }
	     ixOld=ix;
	     xIntOld=xIntercept;
	    }


		    }

     sinovals[jp]=raysum;

	}
__syncthreads();
}

}



